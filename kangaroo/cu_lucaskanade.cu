#include "hip/hip_runtime.h"
#include "cu_lucaskanade.h"

#include <kangaroo/LeastSquareSum.h>
#include <kangaroo/launch_utils.h>

#include <kangaroo/MatUtils.h>

namespace roo
{
__host__ __device__ inline roo::Mat<float, 3, 3> SE2gen(unsigned int genIdx)
{
    roo::Mat<float, 3, 3> gen;
    gen.SetZero();
    switch (genIdx)
    {
    case 0:
        gen(0, 2) = 1;
        break;
    case 1:
        gen(1, 2) = 1;
        break;
    case 2:
        gen(0, 1) = -1;
        gen(1, 0) =  1;
        break;
    }
    return gen;
}

template<typename TO, typename TI>
__global__ void KernWarp(roo::Image<TO> dOutput,
                         roo::Image<TI> dInput,
                         roo::Mat<float, 3, 3> H)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    float3 po_h = make_float3(x, y, 1);
    float3 pi_h = mulSO3(H, po_h);

    float2 po = dn(po_h);
    float2 pi = dn(pi_h);

    if (dInput.InBounds(pi) && dOutput.InBounds(po))
    {
        TI pix = dInput.template GetBilinear<TI>(pi);
        dOutput(po.x, po.y) = ConvertPixel<TO,TI>(pix);
    }
    else
    {
        dOutput(po.x, po.y) = roo::ConvertPixel<TO, float>(0.0f);
    }
}

template<typename TO, typename TI>
void ImageWarp(roo::Image<TO> d_ouput,
               roo::Image<TI> d_input,
               roo::Mat<float, 3, 3> H)
{
    dim3 block;
    dim3 grid;
    roo::InitDimFromOutputImageOver(block, grid, d_ouput, 16, 16);

    KernWarp<TO,TI> <<< grid, block>>>(d_ouput, d_input, H);
    GpuCheckErrors();
}

template<typename TO, typename TI>
__global__ void KernWarp(roo::Image<TO> dOutput,
                         roo::Image<TI> dInput,
                         roo::Image<float> dDepth,
                         roo::Mat<float, 3, 4> T,
                         roo::ImageIntrinsics K)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    float d = dDepth(x, y);

    float2 po   = make_float2(x, y);
    float3 pi_h = roo::mulSE3(T, K.Unproject(po, d));
    float2 pi   = K.Project(pi_h);

    if (dInput.InBounds(pi) && dOutput.InBounds(po))
    {
        TI pix = dInput.template GetBilinear<TI>(pi);
        dOutput(po.x, po.y) = ConvertPixel<TO,TI>(pix);
    }
    else
    {
        dOutput(po.x, po.y) = roo::ConvertPixel<TO, float>(0.0f);
    }
}

template<typename TO, typename TI>
void ImageWarp(roo::Image<TO> d_ouput,
               roo::Image<TI> d_input,
               roo::Image<float> d_depth,
               roo::Mat<float, 3, 4> T,
               roo::ImageIntrinsics K)
{
    dim3 block;
    dim3 grid;
    roo::InitDimFromOutputImageOver(block, grid, d_ouput, 16, 16);

    KernWarp<TO,TI> <<< grid, block>>>(d_ouput, d_input, d_depth, T, K);
    GpuCheckErrors();
}

template<typename TO, unsigned N>
__global__ void KernLucasKanade(roo::Image<TO> dRef,
                                roo::Image<TO> dTemp,
                                roo::Mat<float, 3, 3> H,
                                roo::Image<roo::LeastSquaresSystem<float, N> > dSum)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    float3 pt_h = make_float3(x, y, 1);
    float3 pr_h = roo::mulSO3(H, pt_h);

    float2 pt = roo::dn(pt_h);
    float2 pr = roo::dn(pr_h);

    __shared__ roo::SumLeastSquaresSystem<float, 3, 16, 16> shared_lss;
    roo::LeastSquaresSystem<float, N>& local_lss = shared_lss.ZeroThisObs();

    if (dRef.InBounds(pr, 1) && dTemp.InBounds(pt, 1))
    {
        TO pixRef  = dRef.template GetBilinear<TO>(pr);
        TO pixTemp = dTemp.template GetBilinear<TO>(pt);
        TO pixErr  = pixRef - pixTemp;

        const roo::Mat<float, 2, 3> dPr_by_dpr =
        {
            1.0 / pr_h.z,  0.0,        -pr_h.x / (pr_h.z* pr_h.z),
            0.0,         1.0 / pr_h.z, -pr_h.y / (pr_h.z* pr_h.z)
        };

        roo::Mat<TO, 1, 2>    dIr = dRef.template GetCentralDiff<TO>(pr);
        // from here ignore TO, do type convertion and work on float
        roo::Mat<float, 1, 2> dIr_f;

        dIr_f(0, 0) = roo::ConvertPixel<float, TO>(dIr(0, 0));
        dIr_f(0, 1) = roo::ConvertPixel<float, TO>(dIr(0, 1));

        roo::Mat<float, N, 1> Jr;
        for (int i = 0; i < N; i++)
        {
            Jr(i, 0) = dIr_f * dPr_by_dpr * SE2gen(i) * roo::make_mat(pt_h);
        }

        float err = roo::ConvertPixel<float, TO>(pixErr);
        float w = 1;
        local_lss.JTJ   = roo::OuterProduct(Jr, w);
        local_lss.JTy   = Jr * err * w;
        local_lss.obs   = 1;
        local_lss.sqErr = err * err;
    }

    shared_lss.ReducePutBlock(dSum);
}

template<typename TO>
roo::LeastSquaresSystem<float, 3> LucasKanade(roo::Image<TO> d_reference, roo::Image<TO> d_template, roo::Image<unsigned char> d_workspace, roo::Mat<float, 3, 3> H)
{
    dim3 block;
    dim3 grid;
    roo::InitDimFromOutputImageOver(block, grid, d_template, 16, 16);

    roo::HostSumLeastSquaresSystem<float, 3> global_lss(d_workspace, block, grid);

    KernLucasKanade<TO, 3> <<< grid, block>>>(d_reference, d_template, H, global_lss.LeastSquareImage());
    GpuCheckErrors();
    roo::LeastSquaresSystem<float, 3> lss = global_lss.FinalSystem();

    return lss;
}

// templates instantations
// uchars are not supported as there is no function lerp for them;
// lerp is a basic linear interpolation function required for GetBilinear;
template void ImageWarp<float4,float4>(roo::Image<float4> d_ouput, roo::Image<float4> d_input, roo::Mat<float, 3, 3> H);
template void ImageWarp<float,float>(roo::Image<float> d_ouput, roo::Image<float> d_input, roo::Mat<float, 3, 3> H);
template void ImageWarp<float,float>(roo::Image<float> d_ouput, roo::Image<float> d_input, roo::Image<float> d_depth, roo::Mat<float, 3, 4> T, roo::ImageIntrinsics K);
template void ImageWarp<float,unsigned char>(roo::Image<float> d_ouput, roo::Image<unsigned char> d_input, roo::Image<float> d_depth, roo::Mat<float, 3, 4> T, roo::ImageIntrinsics K);
template void ImageWarp<float,unsigned char>(roo::Image<float> d_ouput, roo::Image<unsigned char> d_input, roo::Mat<float, 3, 3> H);

template roo::LeastSquaresSystem<float, 3> LucasKanade<float>(roo::Image<float> d_reference, roo::Image<float> d_template, roo::Image<unsigned char> d_workspace, roo::Mat<float, 3, 3> H);
template roo::LeastSquaresSystem<float, 3> LucasKanade<float4>(roo::Image<float4> d_reference, roo::Image<float4> d_template, roo::Image<unsigned char> d_workspace, roo::Mat<float, 3, 3> H);
}
